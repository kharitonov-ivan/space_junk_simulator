#include "hip/hip_runtime.h"
#include "model.cuh"
#include <sys/types.h>

void World::World::Save(const std::string& filepath) {
    std::cout << "World::Save not implemented yet\n";
}

void World::World::Load(const std::string& filepath) {
    std::cout << "World::Load not implemented yet\n";
}

void World::World::AddObject(Object& object) {
    objects_.push_back(object);
}

std::vector<std::vector<World::Object> > World::World::CalculatePositions(size_t stepsNumber) {
    std::vector<std::vector<Object> > positions;
    solver_->Solve(dt_, time_, objects_, forces_, stepsNumber, positions);
    return positions;
}

void CheckCollisions(std::vector<std::vector<World::Object> >& positions, std::vector<std::tuple<double, size_t, size_t> >& collisions, double time, double dt) {
    for (size_t i = 0; i < positions[0].size(); ++i) {
        for (size_t j = 0; j < positions.size(); ++j) {
            for (size_t k = j + 1; k < positions.size(); ++k) {
                double dist = pow(positions[j][i].x - positions[k][i].x, 2) + pow(positions[j][i].y - positions[k][i].y, 2) + pow(positions[j][i].z - positions[k][i].z, 2);
                if (dist < pow(positions[j][i].size - positions[k][i].size, 2)) {
                    collisions.push_back(std::make_tuple(time + dt * i, j, k));
                }
            }
        }
    }
}

typedef std::tuple<int, int, int> key_t_;
struct key_hash : public std::unary_function<key_t_, std::size_t>
{
    std::size_t operator()(const key_t_& k) const
    {
        return std::get<0>(k) ^ std::get<1>(k) ^ std::get<2>(k);
    }
};

void CheckCollisionsFast(std::vector<std::vector<World::Object> >& positions, std::vector<std::tuple<double, size_t, size_t> >& collisions, double time, double dt) {
    std::unordered_map<std::tuple<int, int, int>, std::vector<size_t>, key_hash> cubes;
    double size_first = 0.0; 
    double size_second = 0.0;
    for (size_t i = 0; i < positions.size(); ++i) {
        if (positions[i][0].size > size_first) {
            size_second = size_first;
            size_first = positions[i][0].size;
        } else if (positions[i][0].size > size_second) {
            size_second = positions[i][0].size;
        }
    }
    double cube_edge = size_first + size_second;
    for (size_t i = 0; i < positions[0].size(); ++i) {
        cubes.clear();
        for (size_t j = 0; j < positions.size(); ++j) {
            std::tuple<int, int, int> cube = std::make_tuple<int, int, int> (
                trunc(positions[j][i].x / cube_edge),
                trunc(positions[j][i].y / cube_edge),
                trunc(positions[j][i].z / cube_edge)
            );
            cubes[cube].push_back(j);
        }
        for (auto it = cubes.begin(); it != cubes.end(); ++it) {
            auto cur_cube = it->first;
            for (int dx = -1; dx <= 1; ++dx) {
                for (int dy = -1; dy <= 1; ++dy) {
                    for (int dz = -1; dz <= 1; ++dz) {
                        auto next_cube = std::make_tuple<int, int, int>(
                            std::get<0>(cur_cube) + dx,
                            std::get<1>(cur_cube) + dy,
                            std::get<2>(cur_cube) + dz);
                        auto nextIt = cubes.find(next_cube);
                        if (nextIt != cubes.end()) {
                            for (auto nextObj : nextIt->second) {
                                for (auto curObj : it->second) {
                                    if (nextObj < curObj) {
                                        auto nextCoord = positions[nextObj][i];
                                        auto curCoord = positions[curObj][i];
                                        if (pow(nextCoord.x - curCoord.x, 2) + pow(nextCoord.y - curCoord.y, 2) + pow(nextCoord.z - curCoord.z, 2) <
                                            pow(nextCoord.size + curCoord.size, 2)) 
                                        {
                                            collisions.push_back({ time + dt * i, nextObj, curObj });
                                        }
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}

void World::World::Simulate(size_t stepsNumber) {
    size_t curStep = 0;
    while (curStep < stepsNumber) {
        //std::cout << curStep << '\n';
        std::vector<std::vector<Object> > positions = CalculatePositions(
            std::min(maxSteps_, stepsNumber - curStep)
        );
        curStep += maxSteps_;
        //check collisions and log trajectories here

        if (collisionOption_ == "slow") {
            CheckCollisions(positions, collisions_, time_, dt_);
        } else if (collisionOption_ != "disabled") {
            CheckCollisionsFast(positions, collisions_, time_, dt_);
        }

        for (auto it = trajectories_.begin(); it != trajectories_.end(); ++it) {
            it->second.insert(it->second.end(), positions[it->first].begin(), positions[it->first].end());
        }

        for (size_t i = 0; i < positions.size(); ++i) {
            objects_[i] = positions[i][positions[i].size() - 1];
        }

        time_ += dt_ * std::min(maxSteps_, stepsNumber - curStep);
    }
}

void World::World::DumpTrajectories(std::string& outputFile) {
    std::ofstream out(outputFile);
    for (auto it = trajectories_.begin(); it != trajectories_.end(); ++it) {
        out << it->first << '\n';
        for (auto& obj : it->second) {
            out << std::setprecision(20) << obj.x << ' ' << obj.y << ' ' << obj.z << ' ' << obj.vx << ' ' << obj.vy << ' ' << obj.vz << '\n';
        }
    }
}

void World::World::DumpCollisions(std::string& outputFile) {
    std::ofstream out(outputFile);
    for (auto& collision : collisions_) {
        out << std::setprecision(20) << std::get<0>(collision) << ' ' << std::get<1>(collision) << ' ' << std::get<2>(collision) << '\n';
    }
}