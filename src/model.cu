#include "hip/hip_runtime.h"
#include "model.cuh"

void World::World::Save(const std::string& filepath) {
    std::cout << "World::Save not implemented yet\n";
}

void World::World::Load(const std::string& filepath) {
    std::cout << "World::Load not implemented yet\n";
}

void World::World::AddForce(Force& force) {
    forces_.push_back(force);
}

void World::World::AddObject(Object& object) {
    objects_.push_back(object);
}

std::vector<std::vector<World::Object> > World::World::CalculatePositions(size_t stepsNumber) {
    std::vector<std::vector<Object> > positions;
    solver_->Solve(dt_, time_, objects_, forces_, stepsNumber, positions);
    return positions;
}

void CheckCollisions(std::vector<std::vector<World::Object> >& positions, std::vector<std::tuple<float, size_t, size_t> >& collisions, float time, float dt) {
    for (size_t i = 0; i < positions[0].size(); ++i) {
        for (size_t j = 0; j < positions.size(); ++j) {
            for (size_t k = j + 1; k < positions.size(); ++k) {
                float dist = pow(positions[j][i].x - positions[k][i].x, 2) + pow(positions[j][i].y - positions[k][i].y, 2) + pow(positions[j][i].z - positions[k][i].z, 2);
                if (dist < pow(positions[j][i].size - positions[k][i].size, 2)) {
                    collisions.push_back(std::make_tuple(time + dt * i, j, k));
                }
            }
        }
    }
}

void World::World::Simulate(size_t stepsNumber) {
    size_t curStep = 0;
    while (curStep < stepsNumber) {
        //std::cout << curStep << '\n';
        std::vector<std::vector<Object> > positions = CalculatePositions(
            std::min(maxSteps_, stepsNumber - curStep)
        );
        curStep += maxSteps_;

        //check collisions and log trajectories here
        CheckCollisions(positions, collisions_, time_, dt_);

        for (auto& it = trajectories_.begin(); it != trajectories_.end(); ++it) {
            it->second.insert(it->second.end(), positions[it->first].begin(), positions[it->first].end());
        }

        for (size_t i = 0; i < positions.size(); ++i) {
            objects_[i] = positions[i][positions[i].size() - 1];
        }

        time_ += dt_ * std::min(maxSteps_, stepsNumber - curStep);
    }
}

void World::Physics::GravityForce::GetAcceleration(float x, float y, float z,
                                              float vx, float vy, float vz,
                                              float *ax, float *ay, float *az)  {
    float r = sqrt(x * x + y * y + z * z);
    float k = -G * M / r / r / r;
    *ax = x * k;
    *ay = y * k;
    *az = z * k;
};

void World::Force::GetAcceleration(float x, float y, float z, 
                                   float vx, float vy, float vz,
                                   float *ax, float *ay, float *az) {
    *ax = 0.0;
    *ay = 0.0;
    *az = 0.0;
    std::cout << "Warning: GetAcceleration called from base class 'Force'!\n";
}

void World::World::DumpTrajectories(std::string& outputFile) {
    std::ofstream out(outputFile);
    for (auto it = trajectories_.begin(); it != trajectories_.end(); ++it) {
        out << it->first << '\n';
        for (auto& obj : it->second) {
            out << obj.x << ' ' << obj.y << ' ' << obj.z << ' ' << obj.vx << ' ' << obj.vy << ' ' << obj.vz << '\n';
        }
    }
}
