#include "hip/hip_runtime.h"
#include "model.cuh"

void World::World::Save(const std::string& filepath) {
    std::cout << "World::Save not implemented yet\n";
}

void World::World::Load(const std::string& filepath) {
    std::cout << "World::Load not implemented yet\n";
}

void World::World::AddObject(Object& object) {
    objects_.push_back(object);
}

std::vector<std::vector<World::Object> > World::World::CalculatePositions(size_t stepsNumber) {
    std::vector<std::vector<Object> > positions;
    solver_->Solve(dt_, time_, objects_, forces_, stepsNumber, positions);
    return positions;
}

void CheckCollisions(std::vector<std::vector<World::Object> >& positions, std::vector<std::tuple<double, size_t, size_t> >& collisions, double time, double dt) {
    for (size_t i = 0; i < positions[0].size(); ++i) {
        for (size_t j = 0; j < positions.size(); ++j) {
            for (size_t k = j + 1; k < positions.size(); ++k) {
                double dist = pow(positions[j][i].x - positions[k][i].x, 2) + pow(positions[j][i].y - positions[k][i].y, 2) + pow(positions[j][i].z - positions[k][i].z, 2);
                if (dist < pow(positions[j][i].size - positions[k][i].size, 2)) {
                    collisions.push_back(std::make_tuple(time + dt * i, j, k));
                }
            }
        }
    }
}

void World::World::Simulate(size_t stepsNumber) {
    size_t curStep = 0;
    while (curStep < stepsNumber) {
        //std::cout << curStep << '\n';
        std::vector<std::vector<Object> > positions = CalculatePositions(
            std::min(maxSteps_, stepsNumber - curStep)
        );
        curStep += maxSteps_;
        //check collisions and log trajectories here
//        if (collisionOption_ != "disabled") {
//            CheckCollisions(positions, collisions_, time_, dt_);
//        }
        for (auto it = trajectories_.begin(); it != trajectories_.end(); ++it) {
            it->second.insert(it->second.end(), positions[it->first].begin(), positions[it->first].end());
        }

        for (size_t i = 0; i < positions.size(); ++i) {
            objects_[i] = positions[i][positions[i].size() - 1];
        }

        time_ += dt_ * std::min(maxSteps_, stepsNumber - curStep);
    }
}

void World::World::DumpTrajectories(std::string& outputFile) {
    std::ofstream out(outputFile);
    for (auto it = trajectories_.begin(); it != trajectories_.end(); ++it) {
        out << it->first << '\n';
        for (auto& obj : it->second) {
            out << std::setprecision(20) << obj.x << ' ' << obj.y << ' ' << obj.z << ' ' << obj.vx << ' ' << obj.vy << ' ' << obj.vz << '\n';
        }
    }
}

void World::World::DumpCollisions(std::string& outputFile) {
    std::ofstream out(outputFile);
    for (const auto collision : collisions_) {
        out << std::setprecision(20) << std::get<0>(collision) << ' ' << std::get<1>(collision) << ' ' << std::get<2>(collision) << '\n';
    }
}