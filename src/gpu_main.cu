#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <iostream>
#include <iterator>
#include <algorithm>
#include "model.cuh"
#include "gpu/gpu_solver.cuh"

void ReadConfig(std::istream& in,
                std::vector<World::Object>& objects,
                std::vector<size_t>& logTrajectories,
                std::vector<std::string>& forces,
                size_t& maxSteps,
                size_t& steps,
                double& dt,
                double& time,
                std::string& trajectoriesOutputPath,
                std::string& collisionsOutputPath,
                std::string& collisionOption) {
    in >> time >> dt >> maxSteps >> steps;
    size_t objectsNum, logTrajectoriesNum, forcesNum;
    in >> objectsNum;
    for (size_t i = 0; i < objectsNum; ++i) {
        double x, y, z, vx, vy, vz, size;
        in >> x >> y >> z >> vx >> vy >> vz >> size;
        objects.push_back({ x, y, z, vx, vy, vz, size });
    }

    in >> logTrajectoriesNum;
    for (size_t i = 0; i < logTrajectoriesNum; ++i) {
        size_t id;
        in >> id;
        logTrajectories.push_back(id);
    }

    in >> forcesNum;
    for (size_t i = 0; i < forcesNum; ++i) {
        std::string force;
        in >> force;
        forces.push_back(force);
    }

    in >> trajectoriesOutputPath;
    in >> collisionsOutputPath;
    in >> collisionOption;
}
int main(int argc, char *argv[]) {
    using namespace World::Physics;

    std::string trajectoriesOutputPath, collisionsOutputPath, collisionOption;
    std::vector<World::Object> objects;
    std::vector<std::string> forces;
    std::vector<size_t> logTrajectories;
    double time, dt;
    size_t maxSteps, steps;
    auto in = std::ifstream(argv[1], std::ios_base::in);
    ReadConfig(in, objects, logTrajectories,
        forces,
        maxSteps,
        steps,
        dt,
        time,
        trajectoriesOutputPath,
        collisionsOutputPath,
        collisionOption
    );

    World::Solver *generalSolver = new GPUSolver::Solver();

    World::World world(dt, time, objects, forces, generalSolver, maxSteps, logTrajectories, collisionOption);

    world.Simulate(steps);
    world.DumpTrajectories(trajectoriesOutputPath);
    world.DumpCollisions(collisionsOutputPath);
    delete (generalSolver);
}
