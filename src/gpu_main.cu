#include "hip/hip_runtime.h"
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <iostream>
#include <iterator>
#include <algorithm>
#include "model.h"
#include "gpu_solver.cuh"

int main() {
     using namespace World::Physics;

     std::vector<World::Object> objects;
     std::vector<World::Force> forces;

     World::Object start;
     start.x = 0.0;
     start.y = 1.2 * R;
     start.z = 0.0;
     start.vx = sqrt(G * M / (1.2 * R));
     start.vy = 0.0;
     start.vz = 0.0;
     start.size = 1.0;
     objects.push_back(start);


     World::Physics::GravityForce gravity = World::Physics::GravityForce();
     forces.push_back(gravity);

     World::Solver* generalSolver = new GPUSolver::Solver();
     World::World world(1.0, 0.0, objects, forces, generalSolver, 100, std::vector<size_t>());

     std::cout << "Model created. Starting simulations\n";

     world.PrintObject(0);
     for (size_t i = 0; i < 100; ++i) {
         world.Simulate(1);
         std::cout << "Step " << i << ": R = " << world.GetObject(0).R() << ", V = " << world.GetObject(0).V() << ' '
                   << world.GetObject(0).x << ' ' << world.GetObject(0).y << ' ' << world.GetObject(0).z << '\n';
     }
}
