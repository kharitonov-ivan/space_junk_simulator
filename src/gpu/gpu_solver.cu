#include "hip/hip_runtime.h"
#include "gpu_solver.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

__host__ __device__
struct GpuObject {
  __host__ __device__ GpuObject(double x, double y, double z, double vx, double vy, double vz, double size) :
      x(x), y(y), z(z), vx(vx), vy(vy), vz(vz), size(size) {};
  double x, y, z, vx, vy, vz, size;
};

void GetAccel(double x, double y, double z,
              double vx, double vy, double vz,
              double *ax, double *ay, double *az) {
  //TODO: calculate for each force
  using namespace World::Physics;
  //std::cout << "Called GetAccel with: " << x << ' ' << y << ' ' << z << ' ' << vx << ' ' << vy << ' ' << vz << '\n';
  const double r = sqrt(x * x + y * y + z * z);
  //std::cout << r << '\n';
  const double k = -G * M / r / r / r;
  //std::cout << k << '\n';
  *ax = k * x;
  *ay = k * y;
  *az = k * z;
};

World::Object GetNextState(World::Object &obj,
                           double dt,
                           double time) {
  double dt2 = dt / 2.0;
  double dt1 = time + dt;
  double dh = time + dt2;
  //std::cout << "times: " << time << ' ' << dt2 << ' ' << dt1 << ' ' << dh << '\n';
  double ax, ay, az;
  GetAccel(obj.x, obj.y, obj.z, obj.vx, obj.vy, obj.vz, &ax, &ay, &az);

  //std::cout << ax << ' ' << ay << ' ' << az << '\n';

  double kx1 = dt2 * ax;
  double ky1 = dt2 * ay;
  double kz1 = dt2 * az;

  double lx1 = dt2 * obj.vx;
  double ly1 = dt2 * obj.vy;
  double lz1 = dt2 * obj.vz;

  //std::cout << kx1 << ' ' << ky1 << ' ' << kz1 << ' ' << lx1 << ' ' << ly1 << ' ' << lz1 << ' ' << dt2 << ' ' << dt << ' ' << time << '\n';
  GetAccel(obj.x + lx1, obj.y + ly1, obj.z + lz1,
           obj.vx + kx1, obj.vy + ky1, obj.vz + kz1,
           &ax, &ay, &az);

  //std::cout << ax << ' ' << ay << ' ' << az << '\n';

  double kx2 = dt2 * ax;
  double ky2 = dt2 * ay;
  double kz2 = dt2 * az;

  double lx2 = dt2 * (obj.vx + kx1);
  double ly2 = dt2 * (obj.vy + ky1);
  double lz2 = dt2 * (obj.vz + kz1);

  GetAccel(obj.x + lx2, obj.y + ly2, obj.z + lz2,
           obj.vx + kx2, obj.vy + ky2, obj.vz + kz2,
           &ax, &ay, &az);

  double kx3 = dt * ax;
  double ky3 = dt * ay;
  double kz3 = dt * az;

  double lx3 = dt * (obj.vx + kx2);
  double ly3 = dt * (obj.vy + ky2);
  double lz3 = dt * (obj.vz + kz2);

  GetAccel(obj.x + lx3, obj.y + ly3, obj.z + lz3,
           obj.vx + kx3, obj.vy + ky3, obj.vz + kz3,
           &ax, &ay, &az);

  double kx4 = dt2 * ax;
  double ky4 = dt2 * ay;
  double kz4 = dt2 * az;

  double lx4 = dt2 * (obj.vx + kx3);
  double ly4 = dt2 * (obj.vy + ky3);
  double lz4 = dt2 * (obj.vz + kz3);

  return World::Object(
      obj.x + (lx1 + 2.0 * lx2 + lx3 + lx4) / 3.0,
      obj.y + (ly1 + 2.0 * ly2 + ly3 + ly4) / 3.0,
      obj.z + (lz1 + 2.0 * lz2 + lz3 + lz4) / 3.0,
      obj.vx + (kx1 + 2.0 * kx2 + kx3 + kx4) / 3.0,
      obj.vy + (ky1 + 2.0 * ky2 + ky3 + ky4) / 3.0,
      obj.vz + (kz1 + 2.0 * kz2 + kz3 + kz4) / 3.0,
      obj.size);
}

void GpuSolve(double dt, double time,
              thrust::host_vector<World::Object> &objects,
              thrust::host_vector<World::Force> &forces,
              size_t stepsNumber,
              std::vector<std::vector<World::Object> > &positions) {

  positions.resize(objects.size());
  //std::cout << "time: " << time << '\n';
  //std::cout << "dt: " << dt << '\n';
  for (size_t id = 0; id < positions.size(); ++id) {
    positions[id].push_back(objects[id]);
    for (size_t step = 0; step < stepsNumber; ++step) {
      // std::cout << id << ' ' << step << '\n';
      positions[id].push_back(
          GetNextState(positions[id][step], dt, time + step * dt)
      );
    }
  }
}

void GPUSolver::Solver::Solve(double dt,
                              double time,
                              std::vector<World::Object> &objects,
                              std::vector<World::Force> &forces,
                              size_t stepsNumber,
                              std::vector<std::vector<World::Object> > &positions) {
  thrust::host_vector<GpuObject> host_objects;


  for (size_t idx = 0; idx < objects.size(); ++idx) {
    std::cout << objects[idx].x << '\n';
    auto gpu_object = GpuObject(objects[idx].x, objects[idx].y, objects[idx].z,
                                objects[idx].vx, objects[idx].vy, objects[idx].vz,
                                objects[idx].size);
    host_objects.push_back(gpu_object);
  }
  thrust::device_vector<GpuObject> device_objects(host_objects.begin(), host_objects.end());
//  thrust::host_vector<World::Force> host_forces(forces.begin(), forces.end());
//  GpuSolve(dt, time, host_objects, host_forces, stepsNumber, positions);
}