#include "hip/hip_runtime.h"
#include "gpu_solver.cuh"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

struct ForcesOptions {
    //add options for your force here
    bool gravityEnabled, heterogeneousGravityEnabled, airDensityEnabled;
};

__host__ __device__
void GetHeterogeneousGravityAcc(double x, double y, double z,
                                double vx, double vy, double vz,
                                double *ax, double *ay, double *az, ForcesOptions& options) {
    if (!options.heterogeneousGravityEnabled) {
        return;
    }
    double r = sqrt(x * x + y * y + z * z);
    const double pi = 3.14159265359;
    const double EARTH_MASS = 5.972e24;
    const double EARTH_RADIUS = 6.371e6;
    const double GRAVITY_CONSTANT = 6.67408e-11;

    //aliases

    const double M = EARTH_MASS;
    const double G = GRAVITY_CONSTANT;
    const double R = EARTH_RADIUS;


    // TODO Gravity latitude correction (WGS - elipsoid) _ WIP
    /*double a = 6378137.0; //WGS84 Semimajor axis
    double f = 1/298.257223563; //WGS84 Flatteing
    double b = a*(1-f); //Semiminor axis
    double g_equator_const = 9.7803453359;
    double g_pole_const = 9.8321849378;

    double z_wgs = sqrt(b*(1-((x*x+y*y)/a*a))); //Solving for the positive z value on the ellipsoid
    double p = sqrt(x*x+y*y); // Distance from the z-axis
    double wgs_lat = atan(z/(p*(1-f)*p*(1-f)))*180/pi; // Solving the latitude value;

    double eccentricity_squared = 1 - (b /a) * (b /a);
    double k_constant = (b * g_pole_const - a * g_equator_const) / a / g_equator_const;
    double g_corr_lat_num = g_equator_const * (1 + k_constant * sin(wgs_lat) * sin(wgs_lat));
    double g_corr_lat_denum = sqrt(1 - eccentricity_squared * sin(wgs_lat)* sin(wgs_lat));
    double g_corr_wgs = g_corr_lat_num/g_corr_lat_denum;*/

    // TODO Simle circle gravity

    double lat = asin(z / r) * 360 / (2 * pi);

    // Gravity latitude correction (non WGS)
    double g_corr_lat = 9.780327 * (1 + 0.0053024 * sin(lat) * sin(lat) - 0.0000058 * sin(2 * lat) * sin(2 * lat));

    // Gravert altitude correction
    double alt = r - R;
    double g_corr_alt = -3.086e-6 * alt;
    double g_corr = g_corr_lat + g_corr_alt;
    double k = -g_corr / r;

    *ax += x * k;
    *ay += y * k;
    *az += z * k;
};

__host__ __device__
double GpuGetAirDensity(double x, double y, double z,
    double vx, double vy, double vz,
    double *ax, double *ay, double *az, const ForcesOptions& options) {
    if (!options.airDensityEnabled) {
        return 0.0;
    }
    // TODO: Look up table for atmosphere density
    return 0.0;
}

__host__ __device__
void GetSimpleGravityForceAcc(double x, double y, double z,
    double vx, double vy, double vz,
    double *ax, double *ay, double *az, const ForcesOptions& options) {
    if (!options.gravityEnabled) {
        return;
    }
    const double M = 5.972e24;
    const double G = 6.67408e-11;

    double r = sqrt(x * x + y * y + z * z);
    double k = -G * M / r / r / r;
    *ax = x * k;
    *ay = y * k;
    *az = z * k;
};

__host__ __device__
    void GetAccel(double x, double y, double z,
        double vx, double vy, double vz,
        double *ax, double *ay, double *az, const ForcesOptions& options) {
    *ax = 0.0;
    *ay = 0.0;
    *az = 0.0;
    //add your custom force call here
    GetSimpleGravityForceAcc(x, y, z, vx, vy, vz, ax, ay, az, options);
};

__host__ __device__
World::Object GetNextState(const World::Object& obj,
                           double dt,
                           double time,
                           const ForcesOptions& options) {
    double dt2 = dt / 2.0;
    double dt1 = time + dt;
    double dh = time + dt2;
    //std::cout << "times: " << time << ' ' << dt2 << ' ' << dt1 << ' ' << dh << '\n';
    double ax, ay, az;
    GetAccel(obj.x, obj.y, obj.z, obj.vx, obj.vy, obj.vz, &ax, &ay, &az, options);

    //std::cout << ax << ' ' << ay << ' ' << az << '\n';

    double kx1 = dt2 * ax;
    double ky1 = dt2 * ay;
    double kz1 = dt2 * az;

    double lx1 = dt2 * obj.vx;
    double ly1 = dt2 * obj.vy;
    double lz1 = dt2 * obj.vz;

    //std::cout << kx1 << ' ' << ky1 << ' ' << kz1 << ' ' << lx1 << ' ' << ly1 << ' ' << lz1 << ' ' << dt2 << ' ' << dt << ' ' << time << '\n';
    GetAccel(obj.x + lx1, obj.y + ly1, obj.z + lz1,
        obj.vx + kx1, obj.vy + ky1, obj.vz + kz1,
        &ax, &ay, &az, options);

    //std::cout << ax << ' ' << ay << ' ' << az << '\n';

    double kx2 = dt2 * ax;
    double ky2 = dt2 * ay;
    double kz2 = dt2 * az;

    double lx2 = dt2 * (obj.vx + kx1);
    double ly2 = dt2 * (obj.vy + ky1);
    double lz2 = dt2 * (obj.vz + kz1);

    GetAccel(obj.x + lx2, obj.y + ly2, obj.z + lz2,
        obj.vx + kx2, obj.vy + ky2, obj.vz + kz2,
        &ax, &ay, &az, options);

    double kx3 = dt * ax;
    double ky3 = dt * ay;
    double kz3 = dt * az;

    double lx3 = dt * (obj.vx + kx2);
    double ly3 = dt * (obj.vy + ky2);
    double lz3 = dt * (obj.vz + kz2);

    GetAccel(obj.x + lx3, obj.y + ly3, obj.z + lz3,
        obj.vx + kx3, obj.vy + ky3, obj.vz + kz3,
        &ax, &ay, &az, options);

    double kx4 = dt2 * ax;
    double ky4 = dt2 * ay;
    double kz4 = dt2 * az;

    double lx4 = dt2 * (obj.vx + kx3);
    double ly4 = dt2 * (obj.vy + ky3);
    double lz4 = dt2 * (obj.vz + kz3);

    return { obj.x + (lx1 + 2.0 * lx2 + lx3 + lx4) / 3.0,
    obj.y + (ly1 + 2.0 * ly2 + ly3 + ly4) / 3.0,
    obj.z + (lz1 + 2.0 * lz2 + lz3 + lz4) / 3.0,
    obj.vx + (kx1 + 2.0 * kx2 + kx3 + kx4) / 3.0,
    obj.vy + (ky1 + 2.0 * ky2 + ky3 + ky4) / 3.0,
    obj.vz + (kz1 + 2.0 * kz2 + kz3 + kz4) / 3.0,
    obj.size };
}

struct next_pos
    : public thrust::unary_function<World::Object, World::Object> {
        next_pos(double t, double dt, const ForcesOptions& options) : t(t), dt(dt), options(options) {};

    //need constructor with dt and time
    __host__ __device__
        World::Object operator()(const World::Object &x) const {
        //std::unordered_map<double, double> m;
        return GetNextState(x, dt, t, options);
    }
    ForcesOptions options;
    double t, dt;
};

void GpuSolve(double dt, double time,
              thrust::host_vector<World::Object>& objects,
              size_t stepsNumber,
              std::vector<std::vector<World::Object> >& positions,
              const ForcesOptions options) {
    positions.resize(objects.size());

    thrust::device_vector<World::Object> d_objects(objects.begin(), objects.end());
    thrust::device_vector<World::Object> d_next_objects(objects.size());
    thrust::host_vector<World::Object> h_next_objects;
    for (size_t i = 0; i < stepsNumber; ++i) {
        thrust::transform(d_objects.begin(), d_objects.end(), d_next_objects.begin(), next_pos(time, dt, options));
        d_objects = d_next_objects;

        //this section is very slow
        h_next_objects = d_next_objects;
        for (size_t j = 0; j < h_next_objects.size(); ++j) {
            positions[j].push_back(h_next_objects[j]);
        }   
    }
}

ForcesOptions BuildOptions(const std::vector<std::string>& forces) {
    ForcesOptions res;
    for (const auto force : forces) {
        //fill fields you want to pass from config
        if (force == "gravity") {
            res.gravityEnabled = true;
        }
        if (force == "air_density") {
            res.airDensityEnabled = true;
        }
        if (force == "heterogeneous_gravity") {
            res.heterogeneousGravityEnabled = true;
        }
    }
    return res;
}

void GPUSolver::Solver::Solve(
    double dt,
    double time,
    std::vector<World::Object>& objects,
    std::vector<std::string>& forces,
    size_t stepsNumber,
    std::vector<std::vector<World::Object> > &positions) {

    thrust::host_vector<World::Object> host_objects(objects.begin(), objects.end());

    ForcesOptions options = BuildOptions(forces);

    GpuSolve(dt, time, host_objects, stepsNumber, positions, options);
}