#include "hip/hip_runtime.h"
#include "gpu_solver.cuh"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


__host__ __device__
void GetAccel(float x, float y, float z,
    float vx, float vy, float vz,
    float *ax, float *ay, float *az) {
    //TODO: calculate for each force

    const float EARTH_MASS = 5.972e24;
    const float EARTH_RADIUS = 6.371e6;
    const float GRAVITY_CONSTANT = 6.67408e-11;

    //aliases

    const float M = EARTH_MASS;
    const float G = GRAVITY_CONSTANT;
    const float R = EARTH_RADIUS;
    //std::cout << "Called GetAccel with: " << x << ' ' << y << ' ' << z << ' ' << vx << ' ' << vy << ' ' << vz << '\n';
    const float r = sqrt(x * x + y * y + z * z);
    //std::cout << r << '\n';
    const float k = -G * M / r / r / r;
    //std::cout << k << '\n';
    *ax = k * x;
    *ay = k * y;
    *az = k * z;
};

__host__ __device__
World::Object GetNextState(const 
    World::Object& obj,
    float dt,
    float time) {
    float dt2 = dt / 2.0;
    float dt1 = time + dt;
    float dh = time + dt2;
    //std::cout << "times: " << time << ' ' << dt2 << ' ' << dt1 << ' ' << dh << '\n';
    float ax, ay, az;
    GetAccel(obj.x, obj.y, obj.z, obj.vx, obj.vy, obj.vz, &ax, &ay, &az);

    //std::cout << ax << ' ' << ay << ' ' << az << '\n';

    float kx1 = dt2 * ax;
    float ky1 = dt2 * ay;
    float kz1 = dt2 * az;

    float lx1 = dt2 * obj.vx;
    float ly1 = dt2 * obj.vy;
    float lz1 = dt2 * obj.vz;

    //std::cout << kx1 << ' ' << ky1 << ' ' << kz1 << ' ' << lx1 << ' ' << ly1 << ' ' << lz1 << ' ' << dt2 << ' ' << dt << ' ' << time << '\n';
    GetAccel(obj.x + lx1, obj.y + ly1, obj.z + lz1,
        obj.vx + kx1, obj.vy + ky1, obj.vz + kz1,
        &ax, &ay, &az);

    //std::cout << ax << ' ' << ay << ' ' << az << '\n';

    float kx2 = dt2 * ax;
    float ky2 = dt2 * ay;
    float kz2 = dt2 * az;

    float lx2 = dt2 * (obj.vx + kx1);
    float ly2 = dt2 * (obj.vy + ky1);
    float lz2 = dt2 * (obj.vz + kz1);

    GetAccel(obj.x + lx2, obj.y + ly2, obj.z + lz2,
        obj.vx + kx2, obj.vy + ky2, obj.vz + kz2,
        &ax, &ay, &az);

    float kx3 = dt * ax;
    float ky3 = dt * ay;
    float kz3 = dt * az;

    float lx3 = dt * (obj.vx + kx2);
    float ly3 = dt * (obj.vy + ky2);
    float lz3 = dt * (obj.vz + kz2);

    GetAccel(obj.x + lx3, obj.y + ly3, obj.z + lz3,
        obj.vx + kx3, obj.vy + ky3, obj.vz + kz3,
        &ax, &ay, &az);

    float kx4 = dt2 * ax;
    float ky4 = dt2 * ay;
    float kz4 = dt2 * az;

    float lx4 = dt2 * (obj.vx + kx3);
    float ly4 = dt2 * (obj.vy + ky3);
    float lz4 = dt2 * (obj.vz + kz3);

    World::Object res;
    res.x = obj.x + (lx1 + 2.0 * lx2 + lx3 + lx4) / 3.0;
    res.y = obj.y + (ly1 + 2.0 * ly2 + ly3 + ly4) / 3.0;
    res.z = obj.z + (lz1 + 2.0 * lz2 + lz3 + lz4) / 3.0;
    res.vx = obj.vx + (kx1 + 2.0 * kx2 + kx3 + kx4) / 3.0;
    res.vy = obj.vy + (ky1 + 2.0 * ky2 + ky3 + ky4) / 3.0;
    res.vz = obj.vz + (kz1 + 2.0 * kz2 + kz3 + kz4) / 3.0;
    res.size = obj.size;

    return res;
}

struct next_pos
    : public thrust::unary_function<World::Object, World::Object >
{

    next_pos(float t, float dt) : t(t), dt(dt) {};

    //need constructor with dt and time
    __host__ __device__
        World::Object operator()(const World::Object& x) const
    {
        //std::unordered_map<float, float> m;
        return GetNextState(x, 1, 0);

        //return x;
    }

    float t, dt;
};


void GpuSolve(float dt, float time,
    thrust::host_vector<World::Object>& objects,
    thrust::host_vector<World::Force>& forces,
    size_t stepsNumber,
    std::vector<std::vector<World::Object> >& positions) {
    //here come some trash
    positions.resize(objects.size());
 
    thrust::device_vector<World::Object> d_objects(objects.begin(), objects.end());
    thrust::device_vector<World::Object> d_next_objects(objects.size());
    thrust::host_vector<World::Object> h_next_objects;
    for (size_t i = 0; i < stepsNumber; ++i) {
        thrust::transform(d_objects.begin(), d_objects.end(), d_next_objects.begin(), next_pos(0, 1));
        d_objects = d_next_objects;

        //this section is very slow
        h_next_objects = d_next_objects;
        for (size_t j = 0; j < h_next_objects.size(); ++j) {
            positions[j].push_back(h_next_objects[j]);
        }
    }
}

void GPUSolver::Solver::Solve(
    float dt,
    float time,
    std::vector<World::Object>& objects,
    std::vector<World::Force>& forces,
    size_t stepsNumber,
    std::vector<std::vector<World::Object> >& positions) {

    thrust::host_vector<World::Object> host_objects(objects.begin(), objects.end());
    thrust::host_vector<World::Force> host_forces(forces.begin(), forces.end());
    GpuSolve(dt, time, host_objects, host_forces, stepsNumber, positions);    
}